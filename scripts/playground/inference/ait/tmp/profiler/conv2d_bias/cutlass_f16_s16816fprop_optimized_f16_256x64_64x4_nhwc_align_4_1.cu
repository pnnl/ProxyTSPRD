
size_t GLOBAL_WORKSPACE_SIZE = 0;

#include <iostream>
#include <string>
#include <stdexcept>
#include "cutlass/cutlass.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include <cutlass/epilogue/thread/linear_combination_bias_relu.h>
#include <cutlass/epilogue/thread/linear_combination_hardswish.h>



#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_f16_s16816fprop_optimized_f16_256x64_64x4_nhwc_align4"
  using cutlass_tensorop_f16_s16816fprop_optimized_f16_256x64_64x4_nhwc_align4_base = 
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t, 
    cutlass::layout::TensorNHWC,
    cutlass::half_t, 
    cutlass::layout::TensorNHWC,
    cutlass::half_t, 
    cutlass::layout::TensorNHWC,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using DeviceConvFwdInstance = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_f16_s16816fprop_optimized_f16_256x64_64x4_nhwc_align4_base>;



void conv (
    void* in_ptr,
    void* weight_ptr,
    void* out_ptr,
    void* bias_ptr,
    uint8_t* workspace,
    int64_t* batch,
    int64_t* out_ch,
    int64_t* in_ch,
    int64_t* kernel_h,
    int64_t* kernel_w,
    int64_t* in_h,
    int64_t* in_w,
    int64_t* out_batch,
    int64_t* out_h,
    int64_t* out_w,
    int stride,
    int dilation,
    int pad,
    hipStream_t stream
  ) {

  
  int i32_batch = *batch;
  int i32_in_h = *in_h;
  int i32_in_w = *in_w;
  int i32_in_ch = *in_ch;
  int i32_out_ch = *out_ch;
  int i32_kernel_h = *kernel_h;
  int i32_kernel_w = *kernel_w;
  int i32_out_batch = *out_batch;
  int i32_out_h = *out_h;
  int i32_out_w = *out_w;

  using cutlass::layout::TensorNHWC;
  TensorNHWC layout_A(TensorNHWC::packed(cutlass::make_Coord(i32_batch, i32_in_h, i32_in_w, i32_in_ch)));
  TensorNHWC layout_B(TensorNHWC::packed(cutlass::make_Coord(i32_out_ch, i32_kernel_h, i32_kernel_w, i32_in_ch)));
  TensorNHWC layout_C(TensorNHWC::packed(cutlass::make_Coord(i32_out_batch, i32_out_h, i32_out_w, i32_out_ch)));

  cutlass::conv::Conv2dProblemSize problem_size(
    {i32_batch, i32_in_h, i32_in_w, i32_in_ch},
        {i32_out_ch, i32_kernel_h, i32_kernel_w, i32_in_ch},
        {pad, pad, pad, pad},
        {stride, stride},
        {dilation, dilation},
        {i32_out_batch, i32_out_h, i32_out_w, i32_out_ch},
        cutlass::conv::Mode::kCrossCorrelation,
        1
  );

  
  using ElementComputeEpilogue = typename DeviceConvFwdInstance::ElementCompute;
//  TODO: cast to right dtype
  typename DeviceConvFwdInstance::Arguments arguments{
      problem_size,
      {static_cast<cutlass::half_t*>(in_ptr), layout_A},
      {static_cast<cutlass::half_t*>(weight_ptr), layout_B},
      {static_cast<cutlass::half_t*>(bias_ptr), cutlass::layout::TensorNHWC::Stride(0)},
      {static_cast<cutlass::half_t*>(out_ptr), layout_C},
      {ElementComputeEpilogue(1), ElementComputeEpilogue(1)},
  };
  DeviceConvFwdInstance implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> local_workspace(workspace_size);
  workspace = local_workspace.get();
  GLOBAL_WORKSPACE_SIZE = workspace_size;

  auto status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op.initialize(arguments, workspace);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op(stream);
  CUTLASS_CHECK(status);
  return;
  throw std::runtime_error(
      "Unsupported workload for this conv2d specialization."
  );
}

int main(int argc, char** argv) {
  int64_t batch = std::stoi(argv[1]);
  int64_t in_h = std::stoi(argv[2]);
  int64_t in_w = std::stoi(argv[3]);
  int64_t in_ch = std::stoi(argv[4]);
  int64_t kernel_h = std::stoi(argv[5]);
  int64_t kernel_w = std::stoi(argv[6]);
  int64_t out_ch = std::stoi(argv[7]);
  int stride = std::stoi(argv[8]);
  int pad = std::stoi(argv[9]);
  int dilation = std::stoi(argv[10]);
  
  int64_t NI = batch;
  int64_t HI = in_h;
  int64_t WI = in_w;
  int64_t CI = in_ch;
  int64_t CO = out_ch;
  int64_t KH = kernel_h;
  int64_t KW = kernel_w;
  int64_t SH = stride;
  int64_t SW = stride;
  int64_t DH = dilation;
  int64_t DW = dilation;
  int64_t PH = pad;
  int64_t PW = pad;
  int64_t KHEff = (KH - 1) * DH + 1;
  int64_t KWEff = (KW - 1) * DW + 1;
  int64_t NO = NI;
  int64_t HO = (HI + PH + PH - KHEff) / SH + 1;
  int64_t WO = (WI + PW + PW - KWEff) / SW + 1;
  using ElementOutput = typename DeviceConvFwdInstance::ElementC;
  using ElementInputA = typename DeviceConvFwdInstance::ElementA;
  using ElementInputB = typename DeviceConvFwdInstance::ElementB;

  uint8_t* global_workspace = nullptr;
  hipStream_t stream = nullptr;

  cutlass::HostTensor<ElementInputA, typename DeviceConvFwdInstance::LayoutA> x({NI, HI, WI, CI});
  cutlass::HostTensor<ElementInputB, typename DeviceConvFwdInstance::LayoutB> w({CO, KH, KW, CI});
  cutlass::HostTensor<ElementInputB, typename DeviceConvFwdInstance::LayoutB> b({(int)CO, 1, 1, 1});
  cutlass::HostTensor<ElementOutput, typename DeviceConvFwdInstance::LayoutC> y({NO, HO, WO, CO});
  //
  // warmup
  conv(x.device_data(),
       w.device_data(),
       y.device_data(),
       b.device_data(),
       global_workspace,
       &NI,
       &CO,
       &CI,
       &KH,
       &KW,
       &HI,
       &WI,
       &NO,
       &HO,
       &WO,
       stride,
       dilation,
       pad,
       stream);
  hipEvent_t events[2];
  for (auto & event : events) {
    hipEventCreate(&event);
  }
  hipEventRecord(events[0], stream);
  for (int i = 0; i < 5; ++i) {
      conv(x.device_data(),
       w.device_data(),
       y.device_data(),
       b.device_data(),
       global_workspace,
       &NI,
       &CO,
       &CI,
       &KH,
       &KW,
       &HI,
       &WI,
       &NO,
       &HO,
       &WO,
       stride,
       dilation,
       pad,
       stream);
  }
  hipEventRecord(events[1], stream);
  hipEventSynchronize(events[1]);
  float runtime_ms = 0;
  hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }
  // TODO: output workspace
  if (runtime_ms < 0.00001) {
      throw std::runtime_error(
      "OOB in cutlass."
    );
  }
  std::cout << "TIME:" << runtime_ms << std::endl;
  std::cout << "WS:" << GLOBAL_WORKSPACE_SIZE << std::endl;
}
