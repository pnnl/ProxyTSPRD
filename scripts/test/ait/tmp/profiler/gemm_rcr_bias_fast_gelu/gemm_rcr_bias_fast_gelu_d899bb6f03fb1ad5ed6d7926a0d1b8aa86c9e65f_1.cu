#include "hip/hip_runtime.h"

size_t GLOBAL_WORKSPACE_SIZE = 0;


#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <iostream>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/epilogue/thread/linear_combination_silu.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/device_memory.h"


#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"
#include "cutlass/constants.h"
#include "cutlass/complex.h"
#include "cutlass/array.h"
#include "cutlass/half.h"
#include "cutlass/functional.h"
#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/epilogue/thread/linear_combination_generic.h"

namespace cutlass {
namespace epilogue {
namespace thread {

template <
  typename ElementOutput_,                             ///< Data type used to load and store tensors
  int Count,                                           ///< Number of elements computed per operation
                                                       ///< Usually it is 128/sizeof_bits<ElementOutput_>,
                                                       ///< but we use 64 or 32 sometimes when there are not enough data to store
  typename ElementAccumulator_ = ElementOutput_,       ///< Accumulator data type
  typename ElementCompute_ = ElementOutput_,           ///< Data type used to compute linear combination
  ScaleType::Kind Scale = ScaleType::Default,          ///< Control Alpha and Beta scaling
  FloatRoundStyle Round = FloatRoundStyle::round_to_nearest
>
using LinearCombinationFastGELU = LinearCombinationGeneric<GELU_taylor, ElementOutput_, Count, ElementAccumulator_,
                                                          ElementCompute_, Scale, Round, true>;

} // namespace thread
} // namespace epilogue
} // namespace cutlass


#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_0 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_1 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_2 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_3 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_4 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_5 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_6 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    5,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_7 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    6,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_8 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    6,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_9 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    10,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_10 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_11 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_12 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_13 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_14 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_15 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_16 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_17 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_18 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 64>,
    cutlass::gemm::GemmShape<64, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_19 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_20 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombinationFastGELU<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    5,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_21 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8;

template <typename GemmInstance>
void gemm (
    GemmInstance& gemm_op,
    void* a_ptr,
    void* b_ptr,
    void* bias_ptr,
    void* c_ptr,
    uint8_t* workspace,
    int split_k,
    int64_t* a_dim0,
    int64_t* a_dim1,
    int64_t* b_dim0,
    int64_t* b_dim1,
    int64_t* c_dim0,
    int64_t* c_dim1,
  hipStream_t stream
  ) {
  
  int64_t M = (*a_dim0);

  int64_t N = (*b_dim0);

  int64_t K = (*a_dim1);
  
  
  int64_t output_stride = *b_dim0;
  int64_t output_offset = 0;
    
  
  
  int64_t a_size = 1;

    a_size *= *a_dim0;

    a_size *= *a_dim1;

  if (a_size != 0 && !a_ptr) {
    throw std::runtime_error("input a is null!");
  }

  int64_t b_size = 1;

    b_size *= *b_dim0;

    b_size *= *b_dim1;

  if (b_size != 0 && !b_ptr) {
    throw std::runtime_error("input b is null!");
  }

  int64_t c_size = 1;

    c_size *= *c_dim0;

    c_size *= *c_dim1;

  if (c_size != 0) {
    if (!c_ptr) {
      throw std::runtime_error("input c is null!");
    }
  } else {
    // output is empty and safe to return
    return;
  }

  // One of the input tensor are empty
  if (a_size == 0 || b_size == 0) {
    return;
  }

  if (!bias_ptr) {
    throw std::runtime_error("bias_ptr is null!");
  }

  
//  TODO: cast to right dtype
  using ElementComputeEpilogue = typename GemmInstance::ElementAccumulator;

  typename GemmInstance::Arguments arguments{


    cutlass::gemm::GemmUniversalMode::kGemm,
    {M, N, K},
    split_k,
    {ElementComputeEpilogue(1), ElementComputeEpilogue(1)},
    (cutlass::half_t*)(a_ptr),
    (cutlass::half_t*)(b_ptr),
    (cutlass::half_t*)(bias_ptr),
    (cutlass::half_t*)(c_ptr) + output_offset,
    M * K,
    N * K,
    N,
    M * N,
    K,
    K,
    0,
    output_stride

  };

  // https://www.youtube.com/watch?v=rRwxfYlgG-M
  size_t workspace_size = gemm_op.get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> local_workspace(workspace_size);
  workspace = local_workspace.get();
  GLOBAL_WORKSPACE_SIZE = workspace_size;

  auto status = gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = gemm_op.initialize(arguments, workspace, stream);
  CUTLASS_CHECK(status);
  status = gemm_op(stream);
  CUTLASS_CHECK(status);
  return;
      std::cout << "input_ndims0: " << *a_dim0 << std::endl;
      std::cout << "input_ndims1: " << *a_dim1 << std::endl;
      std::cout << "weight_ndims0: " << *b_dim0 << std::endl;
      std::cout << "weight_ndims1: " << *b_dim1 << std::endl;
      std::cout << "output_ndims0: " << *c_dim0 << std::endl;
      std::cout << "output_ndims1: " << *c_dim1 << std::endl;
  throw std::runtime_error(
      "Unsupported workload for this gemm specialization."
  );
}

template <typename GemmInstance>
int benchmark_gemm (


    GemmInstance &gemm_op,
    const char *gemm_op_name,
    void* a_ptr,
    void* b_ptr,

    void* bias_ptr,



    void* c_ptr,
    uint8_t* global_workspace_,

    int split_k,


    int64_t* a_dim0,

    int64_t* a_dim1,


    int64_t* b_dim0,

    int64_t* b_dim1,


    int64_t* c_dim0,

    int64_t* c_dim1,

    hipStream_t stream

  ) {
  // warmup
  for (int i = 0; i < 5; ++i) {
    
{

gemm(

    gemm_op,

    a_ptr,
    b_ptr,

    bias_ptr,

    c_ptr,
    global_workspace_,
    split_k,

    a_dim0,

    a_dim1,


    b_dim0,

    b_dim1,


    c_dim0,

    c_dim1,

    stream
);
}
  }
  hipEvent_t events[2];
  for (auto & event : events) {
    hipEventCreate(&event);
  }
  hipEventRecord(events[0], stream);
  for (int i = 0; i < 10; ++i) {
    
{

gemm(

    gemm_op,

    a_ptr,
    b_ptr,

    bias_ptr,

    c_ptr,
    global_workspace_,
    split_k,

    a_dim0,

    a_dim1,


    b_dim0,

    b_dim1,


    c_dim0,

    c_dim1,

    stream
);
}
  }
  hipEventRecord(events[1], stream);
  hipEventSynchronize(events[1]);
  float runtime_ms = 0;
  hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }
  // TODO: output workspace
  if (runtime_ms < 0.00001) {
      throw std::runtime_error(
      "OOB in cutlass."
    );
  }
  std::cout << "OP:" << gemm_op_name << ",";
  std::cout << "TIME:" << runtime_ms << ",";
  std::cout << "WS:" << GLOBAL_WORKSPACE_SIZE << std::endl;
  return 0;
}

template <typename DType>
struct ProfilerMemoryPool {
  ProfilerMemoryPool() {
    std::random_device rd;
    gen = std::mt19937(rd());
    uniform_dist = std::uniform_int_distribution<int64_t>(1, 48964896);
    offsets.reserve(512);
    strides.reserve(512);
    copies.reserve(512);
    ptrs.reserve(512);
    blobs.reserve(512);
  }
  ~ProfilerMemoryPool() {}

  DType* AllocateGaussianTensor(int64_t size) {
    size_t length = size * sizeof(DType);
    blobs.emplace_back(length);
    DType* ptr = reinterpret_cast<DType*>(blobs.back().get());

    uint64_t seed = uniform_dist(gen);
    double mean = 0.f;
    double std = 1.f;

    cutlass::reference::device::BlockFillRandomGaussian(ptr, size, seed, mean,
                                                        std);

    return ptr;
  }


  int AllocateTensor(int64_t size, int64_t copy) {
    offsets.push_back(0);
    strides.push_back(size);
    copies.push_back(copy);
    auto ptr = AllocateGaussianTensor(size * copy);
    ptrs.push_back(reinterpret_cast<void*>(ptr));
    return ptrs.size() - 1;
  }

  DType* RequestTensorByIdx(int idx) {
    auto copy = copies.at(idx);
    auto offset = offsets.at(idx);
    auto stride = strides.at(idx);
    DType* ptr = reinterpret_cast<DType*>(ptrs.at(idx));
    ptr += offset;
    offset += stride;
    if (offset == copy * stride) {
        offset = 0;
    }
    offsets[idx] = offset;
    return ptr;
  }

  std::vector<int64_t> offsets;
  std::vector<int64_t> strides;
  std::vector<int64_t> copies;
  std::vector<void*> ptrs;
  std::vector<cutlass::DeviceAllocation<uint8_t> > blobs;
  std::mt19937 gen;
  std::uniform_int_distribution<int64_t> uniform_dist;
};


int main(int argc, char** argv) {
  int device_idx;
  hipDeviceProp_t device_properties;
  hipError_t result = hipGetDevice(&device_idx);
  auto memory_pool = std::make_unique<ProfilerMemoryPool<half>>();
  if (result != hipSuccess) {
    throw std::runtime_error("hipGetDevice() API call failed.");
  }

  result = hipGetDeviceProperties(&device_properties, device_idx);

  if (result != hipSuccess) {
    throw std::runtime_error("hipGetDeviceProperties() failed");
  }

  
  int64_t M = std::atoi(argv[1]);
  int64_t N = std::atoi(argv[2]);
  int64_t K = std::atoi(argv[3]);
  int64_t split_k = std::atoi(argv[4]);

  int64_t a_dim0 = M;
  int64_t a_dim1 = K;
  int64_t b_dim0 = N;
  int64_t b_dim1 = K;
  int64_t c_dim0 = M;
  int64_t c_dim1 = N;

  uint8_t* global_workspace_ = nullptr;
  hipStream_t stream = nullptr;

  
  int64_t a_ptr_sz = a_dim0 * a_dim1;
  int64_t b_ptr_sz = b_dim0 * b_dim1;
  int64_t c_ptr_sz = c_dim0 * c_dim1;

  // The value 1 is used to force ptr_max_sz to be non-zero
  int64_t ptr_max_sz = std::max<int64_t>({1, a_ptr_sz, b_ptr_sz, c_ptr_sz});
  // TODO: special pool size for A100 L2 cache 40M
  // need to tune it for other devices
  int64_t mem_pool_sz = std::max(2,  std::min(64, int((1 << 25) / ptr_max_sz)));

  memory_pool->AllocateTensor(a_ptr_sz, mem_pool_sz);  // a_ptr: index 0
  memory_pool->AllocateTensor(b_ptr_sz, mem_pool_sz);  // b_ptr: index 1
  memory_pool->AllocateTensor(c_ptr_sz, mem_pool_sz);  // c_ptr: index 2


  memory_pool->AllocateTensor(c_dim1, mem_pool_sz);  // bias_ptr: index 3



  
  {
  
  GemmInstance_0 gemm_op_0;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_0,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_1 gemm_op_1;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_1,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_2 gemm_op_2;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_2,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_3 gemm_op_3;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_3,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_4 gemm_op_4;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_4,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_5 gemm_op_5;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_5,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_6 gemm_op_6;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_6,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_7 gemm_op_7;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_7,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_8 gemm_op_8;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_8,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_9 gemm_op_9;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_9,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_10 gemm_op_10;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_10,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_11 gemm_op_11;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_11,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_12 gemm_op_12;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_12,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_13 gemm_op_13;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_13,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_14 gemm_op_14;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_14,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_15 gemm_op_15;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_15,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_16 gemm_op_16;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_16,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_17 gemm_op_17;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_17,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_18 gemm_op_18;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_18,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_19 gemm_op_19;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_19,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_20 gemm_op_20;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_20,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_21 gemm_op_21;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_gemm(
      gemm_op_21,
      gemm_op_name,
      memory_pool->RequestTensorByIdx(0),
      memory_pool->RequestTensorByIdx(1),

      memory_pool->RequestTensorByIdx(3),



      memory_pool->RequestTensorByIdx(2),
      global_workspace_,

      split_k,


      &a_dim0,

      &a_dim1,


      &b_dim0,

      &b_dim1,


      &c_dim0,

      &c_dim1,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }
  return 0;
}