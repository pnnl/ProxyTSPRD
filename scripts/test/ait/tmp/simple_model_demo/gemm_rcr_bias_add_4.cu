
#include <iostream>
#include <memory>
#include <random>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/epilogue/thread/linear_combination_residual_block_v2.h"
#include "cutlass/gemm/device/gemm_universal_with_broadcast.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/device_memory.h"

#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8 = 
    cutlass::gemm::device::GemmUniversalWithBroadcast<
        cutlass::half_t, cutlass::layout::RowMajor,
        cutlass::half_t, cutlass::layout::ColumnMajor,
        cutlass::half_t, cutlass::layout::RowMajor,
        float,
        cutlass::arch::OpClassTensorOp,
            cutlass::arch::Sm80,
            cutlass::gemm::GemmShape<128, 64, 64>,
            cutlass::gemm::GemmShape<64, 32, 64>,
            cutlass::gemm::GemmShape<16, 8, 16>,
        cutlass::epilogue::thread::LinearCombinationResidualBlockV2<
            cutlass::half_t, float, float,
            cutlass::half_t,       8,
            cutlass::epilogue::thread::Identity, cutlass::plus, cutlass::epilogue::thread::Identity

        >,
        cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
            3,
            8,
            8
    >;

using f7c3b3594dcebb6b65caaf42904e9fc6ec13219e3 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8;

void gemm_rcr_bias_add_4 (
    void* a_ptr,
    void* b_ptr,
    void* bias_ptr,
    void* d0_ptr,
    void* c_ptr,
    uint8_t* workspace,
    int split_k,
    int64_t* a_dim0,
    int64_t* a_dim1,
    int64_t* b_dim0,
    int64_t* b_dim1,
    int64_t* c_dim0,
    int64_t* c_dim1,
    hipStream_t stream
  ) {
  
 int64_t M = (*a_dim0);

 int64_t N = (*b_dim0);

 int64_t K = (*a_dim1);
  
  int64_t input_a_batch_stride = M * K;
  int64_t input_a_stride = K;
  int64_t input_a_offset = 0; // default to 0
  int64_t input_b_batch_stride = N * K;
  int64_t input_b_stride = K;
  int64_t input_b_offset = 0; // default to 0
    
  
  
  int64_t output_stride = N;
  int64_t output_offset = 0;
  
    
  
  
  int64_t a_size = 1;

    a_size *= *a_dim0;

    a_size *= *a_dim1;

  if (a_size != 0 && !a_ptr) {
    throw std::runtime_error("input a is null!");
  }

  int64_t b_size = 1;

    b_size *= *b_dim0;

    b_size *= *b_dim1;

  if (b_size != 0 && !b_ptr) {
    throw std::runtime_error("input b is null!");
  }

  int64_t c_size = 1;

    c_size *= *c_dim0;

    c_size *= *c_dim1;

  if (c_size != 0) {
    if (!c_ptr) {
      throw std::runtime_error("input c is null!");
    }
  } else {
    // output is empty and safe to return
    return;
  }

  // One of the input tensor are empty
  if (a_size == 0 || b_size == 0) {
    return;
  }

  if (!bias_ptr) {
    throw std::runtime_error("bias is null!");
  }
  if (!d0_ptr) {
    throw std::runtime_error("d0_ptr is null!");
  }

  
  if (M == 1024 && N == 512 && K == 2048) {
    
//  TODO: cast to right dtype
    using ElementComputeEpilogue = typename f7c3b3594dcebb6b65caaf42904e9fc6ec13219e3::ElementAccumulator;

    typename f7c3b3594dcebb6b65caaf42904e9fc6ec13219e3::Arguments arguments{


    cutlass::gemm::GemmUniversalMode::kGemm,
    { M, N, K },

    split_k,

    {ElementComputeEpilogue(1), ElementComputeEpilogue(1)},
    (cutlass::half_t*)(a_ptr) + input_a_offset,
    (cutlass::half_t*)(b_ptr) + input_b_offset,
    (cutlass::half_t*)(d0_ptr),

    nullptr,

    (cutlass::half_t*) (c_ptr) + output_offset,
    (cutlass::half_t*) (bias_ptr),
    nullptr,
    /*batch_stride_A*/ input_a_batch_stride,
    /*batch_stride_B*/ input_b_batch_stride,
    /*batch_stride_C1*/ 0,
    /*batch_stride_C2*/ 0,
    /*batch_stride_D*/ 0,
    /*batch_stride_Vector*/ 0,
    /*batch_stride_Tensor*/ 0,
    input_a_stride,
    input_b_stride,
    N,

    0,

    output_stride,
    /*ldr*/ 0,
    /*/ldt*/ 0

    };

    f7c3b3594dcebb6b65caaf42904e9fc6ec13219e3 gemm_op;

    auto status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);
    status = gemm_op.initialize(arguments, workspace, stream);
    CUTLASS_CHECK(status);
    status = gemm_op(stream);
    CUTLASS_CHECK(status);
    return;
  }
  throw std::runtime_error(
      "Unsupported workload for this gemm_rcr_bias_add_4 specialization."
  );
}